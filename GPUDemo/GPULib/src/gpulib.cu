#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "hip/hip_runtime.h"

extern "C" {
#include "gpulib.h"

#define DEBUG

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


void GPU_GetDevInfo()
{
    int nDevices;
  	hipGetDeviceCount(&nDevices);
	printf("\nGPUs on this system:\n");
  	for (int i = 0; i < nDevices; i++) {
    	hipDeviceProp_t prop;
    	hipGetDeviceProperties(&prop, i);
    	printf("Device Number: %d\n", i);
    	printf("  Device name: %s\n", prop.name);
    	printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    	printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    	printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  	}
}

int GPU_SetDevice(int gpu_dev)
{
    int rv = hipSetDevice(gpu_dev);
    hipDeviceProp_t prop;
    int deviceID;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID);
    printf("The selected GPU Device Info:\r\n");
    printf("%-25s: %d\r\n", "MaxThreadsPerBlock", prop.maxThreadsPerBlock);
    printf("%-25s: %d %d %d\r\n","maxThreadsDim", prop.maxThreadsDim[0], \
                                                  prop.maxThreadsDim[1], \
                                                  prop.maxThreadsDim[2]);
    printf("%-25s: %d %d %d\r\n","maxGridSize",   prop.maxGridSize[0], \
                                                  prop.maxGridSize[1], \
                                                  prop.maxGridSize[2]);
    if(!prop.deviceOverlap)
        return 1;
    else
        return 0;
}

void GPU_MallocBuffer(void *buf, int size)
{
    hipMalloc((void**)&buf, size);
}

void Host_MallocBuffer(void *buf, int size){
    hipHostMalloc((void**)&buf, size);
}

float GPU_MoveDataFromHost(void *src, void *dst, int size)
{
	hipEvent_t startEvent, stopEvent;   
    
    hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	hipEventRecord(startEvent, 0);
	hipError_t(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);

	float time;
	hipEventElapsedTime(&time, startEvent, stopEvent);
	return size * 1e-6 * 8 / time;
}

void GPU_MoveDataToHost(void *src, void *dst, int size)
{
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
    
}

void GPU_FreeBuffer(void *buf)
{
     hipFree(buf);
}
}
