#include <stdio.h>
#include <stdlib.h>

#include "hip/hip_runtime.h"

extern "C" {
#include "gpulib.h"

void GPU_GetDevInfo()
{
    int nDevices;
  	hipGetDeviceCount(&nDevices);
	printf("\nGPUs on this system:\n");
  	for (int i = 0; i < nDevices; i++) {
    	hipDeviceProp_t prop;
    	hipGetDeviceProperties(&prop, i);
    	printf("Device Number: %d\n", i);
    	printf("  Device name: %s\n", prop.name);
    	printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    	printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    	printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  	}
}

int GPU_SetDevice(int gpu_dev)
{
    int rv = hipSetDevice(gpu_dev);
    hipDeviceProp_t prop;
    int deviceID;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID);
    printf("The selected GPU Device Info:\r\n");
    printf("%-25s: %d\r\n", "MaxThreadsPerBlock", prop.maxThreadsPerBlock);
    printf("%-25s: %d %d %d\r\n","maxThreadsDim", prop.maxThreadsDim[0], \
                                                  prop.maxThreadsDim[1], \
                                                  prop.maxThreadsDim[2]);
    printf("%-25s: %d %d %d\r\n","maxGridSize",   prop.maxGridSize[0], \
                                                  prop.maxGridSize[1], \
                                                  prop.maxGridSize[2]);
    if(!prop.deviceOverlap)
        return 1;
    else
        return 0;
}

void GPU_MallocBuffer(void *buf, int size)
{
    hipMalloc((void**)&buf, size);
}

void GPU_MoveDataFromHost(void *src, void *dst, int size)
{
    hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
}

void GPU_MoveDataToHost(void *src, void *dst, int size)
{
    hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);
}

void GPU_FreeBuffer(void *buf)
{
     hipFree(buf);
}
}