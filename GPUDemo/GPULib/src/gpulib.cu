#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#include "hip/hip_runtime.h"

extern "C" {
#include "gpulib.h"

#define DEBUG

hipEvent_t startEvent, stopEvent;
float transfer_time;

inline
hipError_t checkCuda(hipError_t result)
{
#if defined(DEBUG) || defined(_DEBUG)
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
#endif
  return result;
}


void GPU_Init()
{
	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);
}

void GPU_GetDevInfo()
{
    int nDevices;
  	hipGetDeviceCount(&nDevices);
	printf("\nGPUs on this system:\n");
  	for (int i = 0; i < nDevices; i++) {
    	hipDeviceProp_t prop;
    	hipGetDeviceProperties(&prop, i);
    	printf("Device Number: %d\n", i);
    	printf("  Device name: %s\n", prop.name);
    	printf("  Memory Clock Rate (KHz): %d\n", prop.memoryClockRate);
    	printf("  Memory Bus Width (bits): %d\n", prop.memoryBusWidth);
    	printf("  Peak Memory Bandwidth (GB/s): %f\n\n", 2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);
  	}
}

int GPU_SetDevice(int gpu_dev)
{
    int rv = hipSetDevice(gpu_dev);
    hipDeviceProp_t prop;
    int deviceID;
    hipGetDevice(&deviceID);
    hipGetDeviceProperties(&prop, deviceID);
    printf("The selected GPU Device Info:\r\n");
    printf("%-25s: %d\r\n", "MaxThreadsPerBlock", prop.maxThreadsPerBlock);
    printf("%-25s: %d %d %d\r\n","maxThreadsDim", prop.maxThreadsDim[0], \
                                                  prop.maxThreadsDim[1], \
                                                  prop.maxThreadsDim[2]);
    printf("%-25s: %d %d %d\r\n","maxGridSize",   prop.maxGridSize[0], \
                                                  prop.maxGridSize[1], \
                                                  prop.maxGridSize[2]);
    if(!prop.deviceOverlap)
        return 1;
    else
        return 0;
}

void GPU_MallocBuffer(void **buf, uint64_t size)
{
    hipMalloc(buf, size);
}

void Host_MallocBuffer(void **buf, int size){
    int status;
	status = hipHostMalloc(buf, size);
	printf("status: %d\n", status);
}

float GPU_MoveDataFromHost(void *src, void *dst, int size)
{ 
	hipEventRecord(startEvent, 0);
	//hipError_t(hipMemcpy(dst, src, size, hipMemcpyHostToDevice));
	hipMemcpy(dst, src, size, hipMemcpyHostToDevice);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);

	hipEventElapsedTime(&transfer_time, startEvent, stopEvent);
	return size * 1e-6  / transfer_time;
}

void GPU_MoveDataToHost(void *src, void *dst, int size)
{
hipMemcpy(dst, src, size, hipMemcpyDeviceToHost);

}

void GPU_FreeBuffer(void *buf)
{
 hipFree(buf);
}

void Host_FreeBuffer(void *buf)
{
hipHostFree(buf);
}
}

int Host_PinMem(void *buf, int size)
{
return hipHostRegister(buf, size, hipHostRegisterPortable );
}

void Host_UnpinMen(void *buf)
{
	hipHostUnregister(buf);
}
