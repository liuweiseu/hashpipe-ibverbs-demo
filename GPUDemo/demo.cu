#include <stdio.h>
#include <hip/hip_runtime.h>

int main()
{
	void *host_buf, *gpu_buf;
	int size = 8192 * 16384;
	hipEvent_t startEvent, stopEvent;
	float transfer_time, gbps;

	hipEventCreate(&startEvent);
	hipEventCreate(&stopEvent);

	hipHostMalloc((void**)&host_buf, size, hipHostMallocDefault);
	hipMalloc((void **)&gpu_buf, size);

	hipEventRecord(startEvent, 0);
	hipMemcpy(gpu_buf, host_buf, size, hipMemcpyHostToDevice);
	hipEventRecord(stopEvent, 0);
	hipEventSynchronize(stopEvent);
	hipEventElapsedTime(&transfer_time, startEvent, stopEvent);
	gbps = size * 1e-6  / transfer_time;
	printf("transfer_time: %f\n", transfer_time);
	printf("Gbps: %.2f Gbps\n", gbps);

	hipFree(gpu_buf);
	hipHostFree(host_buf);
}
